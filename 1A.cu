#include "hip/hip_runtime.h"
/*------------------------------------------
 ---- Created By: Aditya Avinash Atluri ----
 ------- you are free to use any code ------
 ------- Submit any issues or errors -------
 -------------------------------------------*/

#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include"hip/hip_runtime.h"

__device__ __global__ void Mean1a(float *,float *);
__device__ __global__ void Meanfinald(float *,float *);
__device__ __global__ void Std(float *);
__device__ __global__ void Std1(float *,float *);

#define Block 512  	//Size of Array in Shared Memory and Number of threads in a block
#define Grid 2			//Size of Number of Blocks
#define Total 1024		//Total number of Samples to be processed
#define Max 1024
#define Width 4096		//Total number of Samples per Grid
#define Length 16384	//Total Size of Shared Memory

int main(void){
	float A[Max],B[Max/Block];
	for(int i=0;i<Max;i++){
		A[i]=(i+1)/100;
	}
	for(int i=0;i<(Max/Block);i++){
		B[i]=0;
	}
	float *Ad,*Bd,K=1,*Kd;
	int size=Max*sizeof(float);
	int sizeb=(Max/Block)*sizeof(float);
	int sizek=sizeof(float);
	hipMalloc((void**)&Ad,size);
	hipMalloc((void**)&Bd,sizeb);
	hipMemcpy(Ad,A,size,hipMemcpyHostToDevice);
	hipMemcpy(Bd,B,sizeb,hipMemcpyHostToDevice);
	dim3 dimBlock(Block,1);
	dim3 dimGrid(Grid,1);
	dim3 dimBlock1(1,1);
	dim3 dimGrid1(1,1);
	Mean1a<<<dimGrid,dimBlock>>>(Ad,Bd);
	Meanfinald<<<dimGrid1,dimBlock1>>>(Bd);
	Std<<<dimGrid,dimBlock>>>(Ad,Bd);
	Std1<<<dimGrid1,dimBlock1>>>(Ad,Bd);
	hipMemcpy(&K,Ad,sizek,hipMemcpyDeviceToHost);
	hipMemcpy(B,Bd,sizek,hipMemcpyDeviceToHost);
	printf("%f		%f\n",K,B[0]);
	int quit;
	scanf("%d",&quit);
	return 0;
}


__device__ __global__ void Mean334d(float *Ad,float *Bd){
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	__device__ __shared__ float As[Block];
		As[tx]=Ad[tx+(bx*Block)];
		if(tx%8==0){
			As[tx]=(As[tx]+As[tx+1]+As[tx+2]+As[tx+3]+As[tx+4]+As[tx+5]+As[tx+6]+As[tx+7])/8;
		}
		if(tx%64==0){
			As[tx]=(As[tx]+As[tx+8]+As[tx+16]+As[tx+24]+As[tx+32]+As[tx+40]+As[tx+48]+As[tx+56])/8;
		}
		if(tx==0){
			As[tx]=(As[tx]+As[tx+64]+As[tx+128]+As[tx+192]+As[tx+256]+As[tx+320]+As[tx+384]+As[tx+448])/8;
		}
		Bd[bx]=As[0];
}

__device__ __global__ void Meanfinald(float *Bd){
		Bd[0]=(Bd[0]+Bd[1])/2;
}

__device__ __global__ void Std(float *Ad,float *Bd){
	__device__ __shared__ float As[Block];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	As[tx]=Ad[tx+(bx*Block)];
	As[tx]=As[tx]-Bd[0];
	if(tx%8==0){
			As[tx]=(As[tx]*As[tx])+(As[tx+1]*As[tx+1])+(As[tx+2]*As[tx+2])+(As[tx+3]*As[tx+3])+(As[tx+4]*As[tx+4])+(As[tx+5]*As[tx+5])+(As[tx+6]*As[tx+6])+(As[tx+7]*As[tx+7]);
	}
	if(tx%64==0){
			As[tx]=(As[tx]+As[tx+8]+As[tx+16]+As[tx+24]+As[tx+32]+As[tx+40]+As[tx+48]+As[tx+56]);
	}
	if(tx==0){
		As[tx]=(As[tx]+As[tx+64]+As[tx+128]+As[tx+192]+As[tx+256]+As[tx+320]+As[tx+384]+As[tx+448]);
	}
	Ad[bx]=As[0];
}

__device__ __global__ void Std1(float *Ad,float*Bd){
	Ad[0]=Ad[0]+Ad[1];
	Ad[0]=sqrt(Ad[0]/Max);
}

//	Here, we have a block having 512 threads.
//	Each Grid has 32 Blocks. And, we have only 1 Grid. You know.!!
//	We use shared memory the total 16KB (16384B).
//	We divide it into 32 parts. As we have 32 Blocks.
//	Each Block now has 512B of Memory (128 of floats) (32*4*128)
