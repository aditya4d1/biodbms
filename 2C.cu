#include "hip/hip_runtime.h"
/*------------------------------------------
 ---- Created By: Aditya Avinash Atluri ----
 ------- you are free to use any code ------
 ------- Submit any issues or errors -------
 -------------------------------------------*/

#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include"hip/hip_runtime.h"

__device__ __global__ void Mean2c(float *,float *);
__device__ __global__ void Meanfinald(float *);
__device__ __global__ void Std(float *,float *);
__device__ __global__ void Std1(float *,float *);

#define Block 512  	//Size of Array in Shared Memory and Number of threads in a block
#define Grid 8			//Size of Number of Blocks
#define Total 131072	//Total number of Samples per block
#define Width 4096		//Total number of Samples per Grid
#define Length 16384	//Total Size of Shared Memory
#define Max 1048576		//Total number of samples

int main(void){
	float *A,B[Max/Block];
	A=(float *)malloc(sizeof(float)*Max);
	for(int i=0;i<Max;i++){
		A[i]=i;
	}
	for(int i=0;i<(Max/Block);i++){
		B[i]=0;
	}
	float *Ad,*Bd;
	int size=Max*sizeof(float);
	int sizeb=(Max/Block)*sizeof(float);
	int sizek=sizeof(float);
	int loop=(Max/Width);
	hipMalloc((void**)&Ad,size);
	hipMalloc((void**)&Bd,sizeb);
	hipMemcpy(Ad,A,size,hipMemcpyHostToDevice);
	hipMemcpy(Bd,B,sizeb,hipMemcpyHostToDevice);
	dim3 dimBlock(Block,1);
	dim3 dimGrid(Grid,1);
	dim3 dimBlock1(256,1);
	dim3 dimGrid1(Grid,1);
	dim3 dimBlock2(256,256,8);
	dim3 dimGrid2(8,1);
	for(int i=0;i<loop;i++){
		Mean2c<<<dimGrid,dimBlock>>>(Ad+(i*Width),Bd+(i*Grid));
	}
	Meanfinald<<<dimGrid1,dimBlock1>>>(Bd);
	for(int j=0;j<loop;j++){
		Std<<<dimGrid,dimBlock>>>(Ad+(j*Width),Bd);
	}
	Std1<<<dimGrid2,dimBlock2>>>(Ad,Bd);
	hipMemcpy(B,Ad,8*sizek,hipMemcpyDeviceToHost);
	for(int i=0;i<8;i++){
		printf("%f	\n",B[i]);
	}
	int quit;
	scanf("%d",&quit);
	return 0;
}


__device__ __global__ void Mean2c(float *Ad,float *Bd){
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	__device__ __shared__ float As[Block];
		As[tx]=Ad[tx+(bx*Block)];
		if(tx%8==0){
			As[tx]=(As[tx]+As[tx+1]+As[tx+2]+As[tx+3]+As[tx+4]+As[tx+5]+As[tx+6]+As[tx+7])/8;
		}
		if(tx%64==0){
			As[tx]=(As[tx]+As[tx+8]+As[tx+16]+As[tx+24]+As[tx+32]+As[tx+40]+As[tx+48]+As[tx+56])/8;
		}
		if(tx==0){
			As[tx]=(As[tx]+As[tx+64]+As[tx+128]+As[tx+192]+As[tx+256]+As[tx+320]+As[tx+384]+As[tx+448])/8;
		}
		Bd[bx]=As[0];
}

__device__ __global__ void Meanfinald(float *Bd){
	__device__ __shared__ float Bs[256];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	Bs[tx]=Bd[tx+bx*256];
		if(tx%8==0){
			Bs[tx]=(Bs[tx]+Bs[tx+1]+Bs[tx+2]+Bs[tx+3]+Bs[tx+4]+Bs[tx+5]+Bs[tx+6]+Bs[tx+7])/8;
		}
		if(tx%64==0){
			Bs[tx]=(Bs[tx]+Bs[tx+8]+Bs[tx+16]+Bs[tx+24]+Bs[tx+32]+Bs[tx+40]+Bs[tx+48]+Bs[tx+56])/8;
		}
		if(tx==0){
			Bs[tx]=(Bs[tx]+Bs[tx+64]+Bs[tx+128]+Bs[tx+192])/4;
		}
	Bd[bx]=Bs[0];
}

__device__ __global__ void Std(float *Ad,float *Bd){
	__device__ __shared__ float As[Block];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	As[tx]=Ad[tx+(bx*Block)];
	As[tx]=As[tx]-Bd[0];
	if(tx%8==0){
			As[tx]=(As[tx]*As[tx])+(As[tx+1]*As[tx+1])+(As[tx+2]*As[tx+2])+(As[tx+3]*As[tx+3])+(As[tx+4]*As[tx+4])+(As[tx+5]*As[tx+5])+(As[tx+6]*As[tx+6])+(As[tx+7]*As[tx+7]);
	}
	if(tx%64==0){
			As[tx]=(As[tx]+As[tx+8]+As[tx+16]+As[tx+24]+As[tx+32]+As[tx+40]+As[tx+48]+As[tx+56]);
	}
	if(tx==0){
		As[tx]=(As[tx]+As[tx+64]+As[tx+128]+As[tx+192]+As[tx+256]+As[tx+320]+As[tx+384]+As[tx+448]);
	}
	Ad[bx]=As[0];
}

__device__ __global__ void Std1(float *Ad,float*Bd){
	__device__ __shared__ float As[256];
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int tz=threadIdx.z;
	int bx=blockIdx.x;
	Ad[tz+ty*8]=Ad[tz+ty*4096];
	As[tx]=Ad[tx+bx*256];
		if(tx%8==0){
			As[tx]=(As[tx]+As[tx+1]+As[tx+2]+As[tx+3]+As[tx+4]+As[tx+5]+As[tx+6]+As[tx+7]);
		}
		if(tx%64==0){
			As[tx]=(As[tx]+As[tx+8]+As[tx+16]+As[tx+24]+As[tx+32]+As[tx+40]+As[tx+48]+As[tx+56]);
		}
		if(tx==0){
			As[tx]=(As[tx]+As[tx+64]+As[tx+128]+As[tx+192]);
		}
	Ad[bx]=As[0];
	Ad[bx]=sqrt(Ad[bx]/Total);
}

//	Here, we have a block having 512 threads.
//	Each Grid has 32 Blocks. And, we have only 1 Grid. You know.!!
//	We use shared memory the total 16KB (16384B).
//	We divide it into 32 parts. As we have 32 Blocks.
//	Each Block now has 512B of Memory (128 of floats) (32*4*128)
