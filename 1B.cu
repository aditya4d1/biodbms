#include "hip/hip_runtime.h"
/*------------------------------------------
 ---- Created By: Aditya Avinash Atluri ----
 ------- you are free to use any code ------
 ------- Submit any issues or errors -------
 -------------------------------------------*/
 
#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include"hip/hip_runtime.h"

__device__ __global__ void Mean1b(float *,float *);
__device__ __global__ void Meanfinald(float *);
__device__ __global__ void Std(float *,float *);
__device__ __global__ void Std1(float *,float *);

#define Block 512  	//Size of Array in Shared Memory and Number of threads in a block
#define Grid 8			//Size of Number of Blocks
#define Total 16384		//Total number of Samples per Cluster
#define Max 16384		//Total number of Samples
#define Width 4096		//Total number of Samples per Grid
#define Length 16384	//Total Size of Shared Memory

int main(void){
	float A[Max],B[Max/Block];
	for(int i=0;i<Max;i++){
		A[i]=i;
	}
	for(int i=0;i<(Max/Block);i++){
		B[i]=0;
	}
	float *Ad,*Bd,K=1,*Kd;
	int size=Max*sizeof(float);
	int sizeb=(Max/Block)*sizeof(float);
	int sizek=sizeof(float);
	int loop=(Max/Width);
	hipMalloc((void**)&Ad,size);
	hipMalloc((void**)&Bd,sizeb);
	hipMemcpy(Ad,A,size,hipMemcpyHostToDevice);
	hipMemcpy(Bd,B,sizeb,hipMemcpyHostToDevice);
	dim3 dimBlock(Block,1);
	dim3 dimGrid(Grid,1);
	dim3 dimBlock1(Block,1);
	dim3 dimGrid1(4,1);
	dim3 dimBlock2(32,8,4);
	dim3 dimGrid2(1,1);
	for(int i=0;i<loop;i++){
		Mean1b<<<dimGrid,dimBlock>>>(Ad+(i*Width),Bd+(i*Grid));
	}
	Meanfinald<<<dimGrid1,dimBlock1>>>(Bd);
	for(int j=0;j<loop;j++){
		Std<<<dimGrid,dimBlock>>>(Ad+(j*Width),Bd);
	}
	Std1<<<dimGrid2,dimBlock2>>>(Ad,Bd);
	hipMemcpy(&K,Ad,sizek,hipMemcpyDeviceToHost);
	hipMemcpy(B,Bd,sizek,hipMemcpyDeviceToHost);
	printf("%f		%f\n",K,B[0]);
	int quit;
	scanf("%d",&quit);
	return 0;
}


__device__ __global__ void Mean1b(float *Ad,float *Bd){
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	__device__ __shared__ float As[Block];
		As[tx]=Ad[tx+(bx*Block)];
		if(tx%8==0){
			As[tx]=(As[tx]+As[tx+1]+As[tx+2]+As[tx+3]+As[tx+4]+As[tx+5]+As[tx+6]+As[tx+7])/8;
		}
		if(tx%64==0){
			As[tx]=(As[tx]+As[tx+8]+As[tx+16]+As[tx+24]+As[tx+32]+As[tx+40]+As[tx+48]+As[tx+56])/8;
		}
		if(tx==0){
			As[tx]=(As[tx]+As[tx+64]+As[tx+128]+As[tx+192]+As[tx+256]+As[tx+320]+As[tx+384]+As[tx+448])/8;
		}
		Bd[bx]=As[0];
}

__device__ __global__ void Meanfinald(float *Bd){
	__device__ __shared__ float Bs[Max/Block];
	int tx=threadIdx.x;
		Bs[tx]=Bd[tx];
		if(tx%8==0){
			Bs[tx]=(Bs[tx]+Bs[tx+1]+Bs[tx+2]+Bs[tx+3]+Bs[tx+4]+Bs[tx+5]+Bs[tx+6]+Bs[tx+7])/8;
		}
		if(tx==0){
			Bs[tx]=(Bs[tx]+Bs[tx+8]+Bs[tx+16]+Bs[tx+24])/4;
		}
	Bd[0]=Bs[0];
}

__device__ __global__ void Std(float *Ad,float *Bd){
	__device__ __shared__ float As[Block];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	As[tx]=Ad[tx+(bx*Block)];
	As[tx]=As[tx]-Bd[0];
	if(tx%8==0){
			As[tx]=(As[tx]*As[tx])+(As[tx+1]*As[tx+1])+(As[tx+2]*As[tx+2])+(As[tx+3]*As[tx+3])+(As[tx+4]*As[tx+4])+(As[tx+5]*As[tx+5])+(As[tx+6]*As[tx+6])+(As[tx+7]*As[tx+7]);
	}
	if(tx%64==0){
			As[tx]=(As[tx]+As[tx+8]+As[tx+16]+As[tx+24]+As[tx+32]+As[tx+40]+As[tx+48]+As[tx+56]);
	}
	if(tx==0){
		As[tx]=(As[tx]+As[tx+64]+As[tx+128]+As[tx+192]+As[tx+256]+As[tx+320]+As[tx+384]+As[tx+448]);
	}
	Ad[bx]=As[0];
}

__device__ __global__ void Std1(float *Ad,float*Bd){
	__device__ __shared__ float As[Max/Block];
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int tz=threadIdx.z;
	Ad[ty+tz*4]=Ad[ty+tz*4096];
	As[tx]=Ad[tx];
	if(tx%8==0){
		As[tx]=(As[tx]+As[tx+1]+As[tx+2]+As[tx+3]+As[tx+4]+As[tx+5]+As[tx+6]+As[tx+7]);
	}
	if(tx%64==0){
		As[tx]=(As[tx]+As[tx+8]+As[tx+16]+As[tx+24]+As[tx+32]+As[tx+40]+As[tx+48]+As[tx+56]);
	}
	if(tx==0){
		As[tx]=(As[tx]+As[tx+64]+As[tx+128]+As[tx+192]);
	}
	Ad[0]=As[0];
	Ad[0]=sqrt(Ad[0]/Max);
}

//	Here, we have a block having 512 threads.
//	Each Grid has 32 Blocks. And, we have only 1 Grid. You know.!!
//	We use shared memory the total 16KB (16384B).
//	We divide it into 32 parts. As we have 32 Blocks.
//	Each Block now has 512B of Memory (128 of floats) (32*4*128)
