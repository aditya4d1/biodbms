#include "hip/hip_runtime.h"
/*------------------------------------------
 ---- Created By: Aditya Avinash Atluri ----
 ------- you are free to use any code ------
 ------- Submit any issues or errors -------
 -------------------------------------------*/

#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include"hip/hip_runtime.h"

__device__ __global__ void Mean3a(float *,float *);
__device__ __global__ void Meanfinald(float *);
__device__ __global__ void Std(float *,float *);
__device__ __global__ void Std1(float *);
__device__ __global__ void Std2(float *,float *);

#define Block 512  	//Size of Array in Shared Memory and Number of threads in a block
#define Grid 8			//Size of Number of Blocks
#define Total 1024		//Total number of Samples per cluster
#define Max 131072		//Total number of Samples
#define Width 4096		//Total number of Samples per Grid
#define Length 16384	//Total Size of Shared Memory

int main(void){
	float A[Max],B[Max/Block];
	for(int i=0;i<Max;i++){
		A[i]=(i+1);
	}
	for(int i=0;i<(Max/Block);i++){
		B[i]=0;
	}
	float *Ad,*Bd;
	int size=Max*sizeof(float);
	int sizeb=(Max/Block)*sizeof(float);
	int sizek=sizeof(float);
	hipMalloc((void**)&Ad,size);
	hipMalloc((void**)&Bd,sizeb);
	hipMemcpy(Ad,A,size,hipMemcpyHostToDevice);
	hipMemcpy(Bd,B,sizeb,hipMemcpyHostToDevice);
	dim3 dimBlock(Block,1);
	dim3 dimGrid(Grid,1);
	dim3 dimBlock1(256,1);
	dim3 dimGrid1(1,1);
	dim3 dimBlock2(4,1);
	dim3 dimGrid2(32,1);
	dim3 dimBlock3(128,1);
	dim3 dimGrid3(1,1);
	for(int i=0;i<(Max/Width);i++){
		Mean3a<<<dimGrid,dimBlock>>>(Ad+(i*Width),Bd+(i*Grid));
	}
	Meanfinald<<<dimGrid1,dimBlock1>>>(Bd);
	for(int i=0;i<(Max/Width);i++){
		Std<<<dimGrid,dimBlock>>>(Ad+(i*Width));
	}
	Std1<<<dimGrid2,dimBlock2>>>(Ad);
	Std2<<<dimGrid3,dimBlock3>>>(Ad,Bd);
	hipMemcpy(B,Bd,128*sizek,hipMemcpyDeviceToHost);
	for(int i=0;i<128;i++){
		printf("%f	--  SD	%d\n",B[i],i);
	}
	int quit;
	scanf("%d",&quit);
	return 0;
}


__device__ __global__ void Mean3a(float *Ad,float *Bd){
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	__device__ __shared__ float As[Block];
		As[tx]=Ad[tx+(bx*Block)];
		if(tx%8==0){
			As[tx]=(As[tx]+As[tx+1]+As[tx+2]+As[tx+3]+As[tx+4]+As[tx+5]+As[tx+6]+As[tx+7])/8;
		}
		if(tx%64==0){
			As[tx]=(As[tx]+As[tx+8]+As[tx+16]+As[tx+24]+As[tx+32]+As[tx+40]+As[tx+48]+As[tx+56])/8;
		}
		if(tx==0){
			As[tx]=(As[tx]+As[tx+64]+As[tx+128]+As[tx+192]+As[tx+256]+As[tx+320]+As[tx+384]+As[tx+448])/8;
		}
		Bd[bx]=As[0];
}

__device__ __global__ void Meanfinald(float *Bd){
	int tx=threadIdx.x;
	__device__ __shared__ float Bs[256];
	Bs[tx]=Bd[tx];
	if(tx%2==0){
		Bs[tx/2]=(Bs[tx]+Bs[tx+1])/2;
		Bd[tx/2]=Bs[tx/2];
	}
}

__device__ __global__ void Std(float *Ad,float *Bd){
	__device__ __shared__ float As[Block];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	As[tx]=Ad[tx+(bx*Block)];
	As[tx]=As[tx]-Bd[0];
	if(tx%8==0){
			As[tx]=(As[tx]*As[tx])+(As[tx+1]*As[tx+1])+(As[tx+2]*As[tx+2])+(As[tx+3]*As[tx+3])+(As[tx+4]*As[tx+4])+(As[tx+5]*As[tx+5])+(As[tx+6]*As[tx+6])+(As[tx+7]*As[tx+7]);
	}
	if(tx%64==0){
			As[tx]=(As[tx]+As[tx+8]+As[tx+16]+As[tx+24]+As[tx+32]+As[tx+40]+As[tx+48]+As[tx+56]);
	}
	if(tx==0){
		As[tx]=(As[tx]+As[tx+64]+As[tx+128]+As[tx+192]+As[tx+256]+As[tx+320]+As[tx+384]+As[tx+448]);
	}
	Ad[bx]=As[0];
}

__device__ __global__ void Std1(float *Ad){
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	Ad[tx+bx*4]=(Ad[2*tx+bx*Width]+Ad[2*tx+1+bx*Width]);
}

__device__ __global__ void Std2(float *Ad,float *Bd){
	__device__ __shared__ float As[128],Bs[128];
	int tx=threadIdx.x;
	As[tx]=Ad[tx];
	Bs[tx]=Bd[tx];
	Bs[tx]=sqrt(As[tx]/Total);
	Bd[tx]=Bs[tx];
}

//	Here, we have a block having 512 threads.
//	Each Grid has 32 Blocks. And, we have only 1 Grid. You know.!!
//	We use shared memory the total 16KB (16384B).
//	We divide it into 32 parts. As we have 32 Blocks.
//	Each Block now has 512B of Memory (128 of floats) (32*4*128)
