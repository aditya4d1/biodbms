#include "hip/hip_runtime.h"
/*------------------------------------------
 ---- Created By: Aditya Avinash Atluri ----
 ------- you are free to use any code ------
 ------- Submit any issues or errors -------
 -------------------------------------------*/

#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include"hip/hip_runtime.h"

__device__ __global__ void Mean2a(float *,float *);
__device__ __global__ void Meanfinald(float *);
__device__ __global__ void Std(float *,float *);
__device__ __global__ void Std1(float *,float *);

#define Block 512  	//Size of Array in Shared Memory and Number of threads in a block
#define Grid 8			//Size of Number of Blocks
#define Total 1024		//Total number of Samples per Cluster
#define Max 8192		//Total Samples
#define Width 4096		//Total number of Samples per Grid
#define Length 16384	//Total Size of Shared Memory

int main(void){
	float A[Max],B[Max/Block];
	for(int i=0;i<Max;i++){
		A[i]=(i+1);
	}
	for(int i=0;i<(Max/Block);i++){
		B[i]=0;
	}
	float *Ad,*Bd,K=1,*Kd;
	int size=Max*sizeof(float);
	int sizeb=(Max/Block)*sizeof(float);
	int sizek=sizeof(float);
	hipMalloc((void**)&Ad,size);
	hipMalloc((void**)&Bd,sizeb);
	hipMemcpy(Ad,A,size,hipMemcpyHostToDevice);
	hipMemcpy(Bd,B,sizeb,hipMemcpyHostToDevice);
	dim3 dimBlock(Block,1);
	dim3 dimGrid(Grid,1);
	dim3 dimBlock1(1,1);
	dim3 dimGrid1(16,1);
	for(int i=0;i<(Max/Width);i++){
		Mean2a<<<dimGrid,dimBlock>>>(Ad+(i*Total),Bd+(i*Grid));
	}
	Meanfinald<<<dimGrid1,dimBlock1>>>(Bd);
	for(int i=0;i<(Max/Width);i++){
		Std<<<dimGrid,dimBlock>>>(Ad+i*Width,Bd);
	}
	for(int j=0;j<2;j++){
		Std1<<<dimGrid1,dimBlock1>>>(Ad+j*Width,Bd+(j*Grid));
	}
	hipMemcpy(B,Bd,2*Grid*sizek,hipMemcpyDeviceToHost);
	int quit;
	scanf("%d",&quit);
	return 0;
}


__device__ __global__ void Mean2a(float *Ad,float *Bd){
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	__device__ __shared__ float As[Block];
		As[tx]=Ad[tx+(bx*Block)];
		if(tx%8==0){
			As[tx]=(As[tx]+As[tx+1]+As[tx+2]+As[tx+3]+As[tx+4]+As[tx+5]+As[tx+6]+As[tx+7])/8;
		}
		if(tx%64==0){
			As[tx]=(As[tx]+As[tx+8]+As[tx+16]+As[tx+24]+As[tx+32]+As[tx+40]+As[tx+48]+As[tx+56])/8;
		}
		if(tx==0){
			As[tx]=(As[tx]+As[tx+64]+As[tx+128]+As[tx+192]+As[tx+256]+As[tx+320]+As[tx+384]+As[tx+448])/8;
		}
		Bd[bx]=As[0];
}

__device__ __global__ void Meanfinald(float *Bd){
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	if(bx%2==0){
		Bd[bx]=(Bd[bx]+Bd[bx+1])/2;
	}
}

__device__ __global__ void Std(float *Ad,float *Bd){
	__device__ __shared__ float As[Block];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	As[tx]=Ad[tx+(bx*Block)];
	As[tx]=As[tx]-Bd[0];
	if(tx%8==0){
			As[tx]=(As[tx]*As[tx])+(As[tx+1]*As[tx+1])+(As[tx+2]*As[tx+2])+(As[tx+3]*As[tx+3])+(As[tx+4]*As[tx+4])+(As[tx+5]*As[tx+5])+(As[tx+6]*As[tx+6])+(As[tx+7]*As[tx+7]);
	}
	if(tx%64==0){
			As[tx]=(As[tx]+As[tx+8]+As[tx+16]+As[tx+24]+As[tx+32]+As[tx+40]+As[tx+48]+As[tx+56]);
	}
	if(tx==0){
		As[tx]=(As[tx]+As[tx+64]+As[tx+128]+As[tx+192]+As[tx+256]+As[tx+320]+As[tx+384]+As[tx+448]);
	}
	Ad[bx]=As[0];
}

__device__ __global__ void Std1(float *Ad,float*Bd){
	int bx=blockIdx.x;
	if(bx%2==0){
		Ad[bx]=Ad[bx]+Ad[bx+1];
	}
	Bd[bx]=sqrt(Ad[bx]/Total);
}

//	Here, we have a block having 512 threads.
//	Each Grid has 32 Blocks. And, we have only 1 Grid. You know.!!
//	We use shared memory the total 16KB (16384B).
//	We divide it into 32 parts. As we have 32 Blocks.
//	Each Block now has 512B of Memory (128 of floats) (32*4*128)
