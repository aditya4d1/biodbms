#include "hip/hip_runtime.h"
/*------------------------------------------
 ---- Created By: Aditya Avinash Atluri ----
 ------- you are free to use any code ------
 ------- Submit any issues or errors -------
 -------------------------------------------*/

#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include"hip/hip_runtime.h"

__device__ __global__ void Mean3c(float *,float *);
__device__ __global__ void Meanfinald(float *);
__device__ __global__ void Meanfinal1d(float *);
__device__ __global__ void Std(float *,float *);
__device__ __global__ void Std1(float *);
__device__ __global__ void Std2(float *);
__device__ __global__ void Std3(float *,float *);

#define Block 512  	//Size of Array in Shared Memory and Number of threads in a block
#define Grid 8			//Size of Number of Blocks
#define Total 131072	//Total number of Samples per cluster
#define Max 16777216	//Total number of Samples
#define Width 4096		//Total number of Samples per Grid
#define Length 16384	//Total Size of Shared Memory
#define Dimen 128		//Total number of Clusters

int main(void){
	float *A,B[Max/Block];
	(float*)A=(float *)malloc(Max*sizeof(float));
	for(int i=0;i<Max;i++){
		A[i]=(i+1)*0.001;
	}
	for(int i=0;i<(Max/Block);i++){
		B[i]=0;
	}
	float *Ad,*Bd;
	int size=Max*sizeof(float);
	int sizeb=(Max/Block)*sizeof(float);
	int sizek=sizeof(float);
	int loop=(Max/Width);
	hipMalloc((void**)&Ad,size);
	hipMalloc((void**)&Bd,sizeb);
	hipMemcpy(Ad,A,size,hipMemcpyHostToDevice);
	hipMemcpy(Bd,B,sizeb,hipMemcpyHostToDevice);
	dim3 dimBlock(Block,1);
	dim3 dimGrid(Grid,1);
	dim3 dimBlock1(256,1);
	dim3 dimGrid1(16,1);
	dim3 dimBlock2(16,1);
	dim3 dimGrid2(8,1);
	dim3 dimBlock3(8,1);
	dim3 dimGrid3(512,1);
	dim3 dimBlock4(32,1);
	dim3 dimGrid4(128,1);
	dim3 dimBlock5(1,1);
	dim3 dimGrid5(128,1);
	for(int i=0;i<loop;i++){
		Mean3c<<<dimGrid,dimBlock>>>(Ad+(i*Width),Bd+(i*Grid));
	}
	for(int i=0;i<8;i++){
		Meanfinald<<<dimGrid1,dimBlock1>>>(Bd+(i*Width));
	}
	Meanfinal1d<<<dimGrid2,dimBlock2>>>(Bd);
	for(int j=0;j<loop;j++){
		Std<<<dimGrid,dimBlock>>>(Ad+(j*Width));
	}
	for(int i=0;i<8;i++){
		Std1<<<dimGrid3,dimBlock3>>>(Ad+i*2097152);
	}
	Std2<<<dimGrid4,dimBlock4>>>(Ad);
	Std3<<<dimGrid5,dimBlock5>>>(Ad,Bd);
	hipMemcpy(A,Ad,128*sizek,hipMemcpyDeviceToHost);
	for(int i=0;i<128;i++){
		printf("%f	--	%d\n",A[i],i);
	}
	int quit;
	scanf("%d",&quit);
	return 0;
}

__device__ __global__ void Mean3c(float *Ad,float *Bd){
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	__device__ __shared__ float As[Block];
		As[tx]=Ad[tx+(bx*Block)];
		if(tx%8==0){
			As[tx]=(As[tx]+As[tx+1]+As[tx+2]+As[tx+3]+As[tx+4]+As[tx+5]+As[tx+6]+As[tx+7])/8;
		}
		if(tx%64==0){
			As[tx]=(As[tx]+As[tx+8]+As[tx+16]+As[tx+24]+As[tx+32]+As[tx+40]+As[tx+48]+As[tx+56])/8;
		}
		if(tx==0){
			As[tx]=(As[tx]+As[tx+64]+As[tx+128]+As[tx+192]+As[tx+256]+As[tx+320]+As[tx+384]+As[tx+448])/8;
		}
		Bd[bx]=As[0];
}

__device__ __global__ void Meanfinald(float *Bd){
	__device__ __shared__ float Bs[Block/2];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
		Bs[tx]=Bd[tx+bx*256];
		if(tx%8==0){
			Bs[tx]=(Bs[tx]+Bs[tx+1]+Bs[tx+2]+Bs[tx+3]+Bs[tx+4]+Bs[tx+5]+Bs[tx+6]+Bs[tx+7])/8;
		}
		if(tx%64==0){
			Bs[tx]=(Bs[tx]+Bs[tx+8]+Bs[tx+16]+Bs[tx+24]+Bs[tx+32]+Bs[tx+40]+Bs[tx+48]+Bs[tx+56])/8;
		}
		if(tx==0){
			Bs[tx]=(Bs[tx]+Bs[tx+64]+Bs[tx+128]+Bs[tx+192])/4;
		}
	Bd[bx]=Bs[0];
}

__device__ __global__ void Meanfinal1d(float *Bd){
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	Bd[tx+bx*16]=Bd[tx+bx*4096];
}

__device__ __global__ void Std(float *Ad,float *Bd){
	__device__ __shared__ float As[Block];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	As[tx]=Ad[tx+(bx*Block)];
	As[tx]=As[tx]-Bd[0];
	if(tx%8==0){
			As[tx]=(As[tx]*As[tx])+(As[tx+1]*As[tx+1])+(As[tx+2]*As[tx+2])+(As[tx+3]*As[tx+3])+(As[tx+4]*As[tx+4])+(As[tx+5]*As[tx+5])+(As[tx+6]*As[tx+6])+(As[tx+7]*As[tx+7]);
	}
	if(tx%64==0){
			As[tx]=(As[tx]+As[tx+8]+As[tx+16]+As[tx+24]+As[tx+32]+As[tx+40]+As[tx+48]+As[tx+56]);
	}
	if(tx==0){
		As[tx]=(As[tx]+As[tx+64]+As[tx+128]+As[tx+192]+As[tx+256]+As[tx+320]+As[tx+384]+As[tx+448]);
	}
	Ad[bx]=As[0];
}

__device__ __global__ void Std1(float *Ad){
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	__device__ __shared__ float As[8];
	As[tx+tx*8]=Ad[tx+bx*4096];
	if(tx%8==0){
		As[tx]=(As[tx]+As[tx+1]+As[tx+2]+As[tx+3]+As[tx+4]+As[tx+5]+As[tx+6]+As[tx+7]);
	}
	Ad[bx]=As[0];
}

__device__ __global__ void Std2(float *Ad){
	__device__ __shared__ float As[32];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	As[tx]=Ad[tx*4096+bx*131072];
	if(tx%8==0){
		As[tx]=As[tx]+As[tx+1]+As[tx+2]+As[tx+3]+As[tx+4]+As[tx+5]+As[tx+6]+As[tx+7];
	}
	if(tx%32==0){
		As[tx]=As[tx]+As[tx+8]+As[tx+16]+As[tx+24];
	}
	Ad[bx]=As[0];
}

__device__ __global__ void Std3(float *Ad,float *Bd){
	__device__ __shared__ float As[1],Bs[1];
	int bx=blockIdx.x;
	As[0]=Ad[bx];
	Bs[0]=Bd[bx];
	As[0]=sqrt(As[0]/Total));
	Ad[bx]=As[0];
}

//	Here, we have a block having 512 threads.
//	Each Grid has 32 Blocks. And, we have only 1 Grid. You know.!!
//	We use shared memory the total 16KB (16384B).
//	We divide it into 32 parts. As we have 32 Blocks.
//	Each Block now has 512B of Memory (128 of floats) (32*4*128)
