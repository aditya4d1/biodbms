#include "hip/hip_runtime.h"
/*------------------------------------------
 ---- Created By: Aditya Avinash Atluri ----
 ------- you are free to use any code ------
 ------- Submit any issues or errors -------
 -------------------------------------------*/

#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include"hip/hip_runtime.h"

__device__ __global__ void Mean2d(float *,float *);
__device__ __global__ void Meanfinald(float *);
__device__ __global__ void Meanfinal2d(float *);
__device__ __global__ void Std(float *,float *);
__device__ __global__ void Std1(float *);
__device__ __global__ void Std2(float *);
__device__ __global__ void Std3(float *,float *);

#define Block 512  	//Size of Array in Shared Memory and Number of threads in a block
#define Grid 8			//Size of Number of Blocks
#define Total 1048576	//Total number of Samples to be processed
#define Width 4096		//Total number of Samples per Grid
#define Length 16384	//Total Size of Shared Memory
#define Max 8388608		//Total number of Samples
#define Dimen 8			//Total number of Clusters

int main(void){
	float B[Max/Block];
	float *A;
	(float *)A=(float *)malloc(sizeof(float)*Max);
	for(int i=0;i<Max;i++){
		A[i]=(i+1)*0.00001;
	}
	for(int i=0;i<(Max/Block);i++){
		B[i]=0;
	}
	float *Ad,*Bd;
	int size=Max*sizeof(float);
	int sizeb=(Max/Block)*sizeof(float);
	int sizek=sizeof(float);
	int loop=(Max/Width);
	hipMalloc((void**)&Ad,size);
	hipMalloc((void**)&Bd,sizeb);
	hipMemcpy(Ad,A,size,hipMemcpyHostToDevice);
	hipMemcpy(Bd,B,sizeb,hipMemcpyHostToDevice);
	dim3 dimBlock(Block,1);
	dim3 dimGrid(Grid,1);
	dim3 dimBlock1(Block,1);
	dim3 dimGrid1(Grid/2,1);
	dim3 dimBlock2(4,1);
	dim3 dimGrid2(8,1);
	dim3 dimBlock3(8,8);
	dim3 dimGrid3(256,1);
	dim3 dimBlock4(4,1);
	dim3 dimGrid4(8,1);
	for(int i=0;i<loop;i++){
		Mean2d<<<dimGrid,dimBlock>>>(Ad+(i*Width),Bd+(i*Grid));
	}
	for(int i=0;i<8;i++){
		Meanfinald<<<dimGrid1,dimBlock1>>>(Bd+i*(Width/2));
	}
	Meanfinal2d<<<dimGrid2,dimBlock2>>>(Bd);
	for(int j=0;j<loop;j++){
		Std<<<dimGrid,dimBlock>>>(Ad+(j*Width),Bd);
	}
	Std1<<<dimGrid3,dimBlock3>>>(Ad);
	Std2<<<dimGrid1,dimBlock1>>>(Ad+(i*2048));
	Std3<<<dimGrid4,dimBlock4>>>(Ad,Bd);
	hipMemcpy(A,Ad,8*sizek,hipMemcpyDeviceToHost);
	int quit;
	scanf("%d",&quit);
	return 0;
}


__device__ __global__ void Mean2d(float *Ad,float *Bd){
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	__device__ __shared__ float As[Block];
		As[tx]=Ad[tx+(bx*Block)];
		if(tx%8==0){
			As[tx]=(As[tx]+As[tx+1]+As[tx+2]+As[tx+3]+As[tx+4]+As[tx+5]+As[tx+6]+As[tx+7])/8;
		}
		if(tx%64==0){
			As[tx]=(As[tx]+As[tx+8]+As[tx+16]+As[tx+24]+As[tx+32]+As[tx+40]+As[tx+48]+As[tx+56])/8;
		}
		if(tx==0){
			As[tx]=(As[tx]+As[tx+64]+As[tx+128]+As[tx+192]+As[tx+256]+As[tx+320]+As[tx+384]+As[tx+448])/8;
		}
		Bd[bx]=As[0];
}

__device__ __global__ void Meanfinald(float *Bd){
	__device__ __shared__ float Bs[Block];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	Bs[tx]=Bd[tx+bx*Block];
		if(tx%8==0){
			Bs[tx]=(Bs[tx]+Bs[tx+1]+Bs[tx+2]+Bs[tx+3]+Bs[tx+4]+Bs[tx+5]+Bs[tx+6]+Bs[tx+7])/8;
		}
		if(tx%64==0){
			Bs[tx]=(Bs[tx]+Bs[tx+8]+Bs[tx+16]+Bs[tx+24]+Bs[tx+32]+Bs[tx+40]+Bs[tx+48]+Bs[tx+56])/8;
		}
		if(tx==0){
			Bs[tx]=(Bs[tx]+Bs[tx+64]+Bs[tx+128]+Bs[tx+192])/4;
		}
	Bd[bx]=Bs[0];
}

__device__ __global__ void Meanfinal2d(float *Bd){
	__device__ __shared__ float Bs[Block];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	Bs[tx]=Bd[tx+bx*(Width/2)];
	if(tx==0){
		Bs[tx]=(Bs[tx]+Bs[tx+1]+Bs[tx+2]+Bs[tx+3])/4;
	}
	Bd[bx]=Bs[0];
}

__device__ __global__ void Std(float *Ad,float *Bd){
	__device__ __shared__ float As[Block];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	As[tx]=Ad[tx+(bx*Block)];
	As[tx]=As[tx]-Bd[0];
	if(tx%8==0){
		As[tx]=(As[tx]*As[tx])+(As[tx+1]*As[tx+1])+(As[tx+2]*As[tx+2])+(As[tx+3]*As[tx+3])+(As[tx+4]*As[tx+4])+(As[tx+5]*As[tx+5])+(As[tx+6]*As[tx+6])+(As[tx+7]*As[tx+7]);
	}
	if(tx%64==0){
			As[tx]=(As[tx]+As[tx+8]+As[tx+16]+As[tx+24]+As[tx+32]+As[tx+40]+As[tx+48]+As[tx+56]);
	}
	if(tx==0){
		As[tx]=(As[tx]+As[tx+64]+As[tx+128]+As[tx+192]+As[tx+256]+As[tx+320]+As[tx+384]+As[tx+448]);
	}
	Ad[bx]=As[0];
}

__device__ __global__ void Std1(float *Ad){
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int bx=blockIdx.x;
	Ad[tx+bx*8+ty*2048]=Ad[tx+bx*Width+ty*Total];
}

__device__ __global__ void Std2(float *Ad){
	__device__ __shared__ float As[Block];
	int bx=blockIdx.x;
	int tx=threadIdx.x;
	As[tx]=Ad[tx+(bx*Block)];
	if(tx%8==0){
		As[tx]=(As[tx]+As[tx+1]+As[tx+2]+As[tx+3]+As[tx+4]+As[tx+5]+As[tx+6]+As[tx+7]);
	}
	if(tx%64==0){
			As[tx]=(As[tx]+As[tx+8]+As[tx+16]+As[tx+24]+As[tx+32]+As[tx+40]+As[tx+48]+As[tx+56]);
	}
	if(tx==0){
		As[tx]=(As[tx]+As[tx+64]+As[tx+128]+As[tx+192]+As[tx+256]+As[tx+320]+As[tx+384]+As[tx+448]);
	}
	Ad[bx]=As[0];
}

__device__ __global__ void Std3(float *Ad,float *Bd){
	__device__ __shared__ float As[4],Bs[1];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	As[tx]=Ad[tx+bx*2048];
	Bs[0]=Bd[bx];
	if(tx%4==0){
		As[tx]=(As[tx]+As[tx+1]+As[tx+2]+As[tx+3]);
	}
	As[0]=sqrt(As[0]/Total);
	Ad[bx]=As[0];
}


//	Here, we have a block having 512 threads.
//	Each Grid has 32 Blocks. And, we have only 1 Grid. You know.!!
//	We use shared memory the total 16KB (16384B).
//	We divide it into 32 parts. As we have 32 Blocks.
//	Each Block now has 512B of Memory (128 of floats) (32*4*128)
