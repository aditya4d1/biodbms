#include "hip/hip_runtime.h"
/*------------------------------------------
 ---- Created By: Aditya Avinash Atluri ----
 ------- you are free to use any code ------
 ------- Submit any issues or errors -------
 -------------------------------------------*/

#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include"hip/hip_runtime.h"

__device__ __global__ void Mean2b(float *,float *);
__device__ __global__ void Meanfinald(float *);
__device__ __global__ void Std(float *,float *);
__device__ __global__ void Std1(float *);
__device__ __global__ void Std2(float *,float *);

#define Block 512  	//Size of Array in Shared Memory and Number of threads in a block
#define Grid 8			//Size of Number of Blocks
#define Total 16384		//Total number of Samples per Cluster
#define Max 131072		//Total number of Samples
#define Width 4096		//Total number of Samples per Grid
#define Length 16384	//Total Size of Shared Memory

int main(void){
	float A[Max],B[Max/Block];
	for(int i=0;i<Max;i++){
		A[i]=(0.01)*i;
	}
	for(int i=0;i<(Max/Block);i++){
		B[i]=0;
	}
	float *Ad,*Bd,K=1,*Kd;
	int size=Max*sizeof(float);
	int sizeb=(Max/Block)*sizeof(float);
	int sizek=sizeof(float);
	int loop=(Max/Width);
	hipMalloc((void**)&Ad,size);
	hipMalloc((void**)&Bd,sizeb);
	hipMemcpy(Ad,A,size,hipMemcpyHostToDevice);
	hipMemcpy(Bd,B,sizeb,hipMemcpyHostToDevice);
	dim3 dimBlock(Block,1);
	dim3 dimGrid(Grid,1);
	dim3 dimBlock1(Total/Block,1);
	dim3 dimGrid1(8,1);
	for(int i=0;i<loop;i++){
		Mean2b<<<dimGrid,dimBlock>>>(Ad+(i*Width),Bd+(i*Grid));
	}
	Meanfinald<<<dimGrid1,dimBlock1>>>(Bd);
	for(int j=0;j<loop;j++){
		Std<<<dimGrid,dimBlock>>>(Ad+(j*Width));
	}
	Std1<<<dimGrid1,dimBlock1>>>(Ad);
	Std2<<<dimGrid1,dimBlock1>>>(Ad,Bd);
	hipMemcpy(A,Ad,8*sizek,hipMemcpyDeviceToHost);
	int quit;
	scanf("%d",&quit);
	return 0;
}


__device__ __global__ void Mean2b(float *Ad,float *Bd){
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	__device__ __shared__ float As[Block];
		As[tx]=Ad[tx+(bx*Block)];
		if(tx%8==0){
			As[tx]=(As[tx]+As[tx+1]+As[tx+2]+As[tx+3]+As[tx+4]+As[tx+5]+As[tx+6]+As[tx+7])/8;
		}
		if(tx%64==0){
			As[tx]=(As[tx]+As[tx+8]+As[tx+16]+As[tx+24]+As[tx+32]+As[tx+40]+As[tx+48]+As[tx+56])/8;
		}
		if(tx==0){
			As[tx]=(As[tx]+As[tx+64]+As[tx+128]+As[tx+192]+As[tx+256]+As[tx+320]+As[tx+384]+As[tx+448])/8;
		}
		Bd[bx]=As[0];
}


__device__ __global__ void Meanfinald(float *Bd){
	__device__ __shared__ float Bs[Total/Block];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	Bs[tx]=Bd[tx+bx*32];
	if(tx%8==0){
		Bs[tx]=(Bs[tx]+Bs[tx+1]+Bs[tx+2]+Bs[tx+3]+Bs[tx+4]+Bs[tx+5]+Bs[tx+6]+Bs[tx+7])/8;
	}
	if(tx%32==0){
		Bs[tx]=(Bs[tx]+Bs[tx+8]+Bs[tx+16]+Bs[tx+24])/4;
	}
	Bd[bx]=Bs[0];
}

__device__ __global__ void Std(float *Ad,float *Bd){
	__device__ __shared__ float As[Block];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	As[tx]=Ad[tx+(bx*Block)];
	As[tx]=As[tx]-Bd[0];
	if(tx%8==0){
			As[tx]=(As[tx]*As[tx])+(As[tx+1]*As[tx+1])+(As[tx+2]*As[tx+2])+(As[tx+3]*As[tx+3])+(As[tx+4]*As[tx+4])+(As[tx+5]*As[tx+5])+(As[tx+6]*As[tx+6])+(As[tx+7]*As[tx+7]);
	}
	if(tx%64==0){
			As[tx]=(As[tx]+As[tx+8]+As[tx+16]+As[tx+24]+As[tx+32]+As[tx+40]+As[tx+48]+As[tx+56]);
	}
	if(tx==0){
		As[tx]=(As[tx]+As[tx+64]+As[tx+128]+As[tx+192]+As[tx+256]+As[tx+320]+As[tx+384]+As[tx+448]);
	}
	Ad[bx]=As[0];
}


__device__ __global__ void Std1(float *Ad){
	__device__ __shared__ float As[32];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	Ad[bx+tx*8]=Ad[bx+tx*Width];
	As[tx]=Ad[tx+bx*8];
	if(tx%8==0){
		As[tx]=As[tx]+As[tx+1]+As[tx+2]+As[tx+3]+As[tx+4]+As[tx+5]+As[tx+6]+As[tx+7];
	}
	if(tx%32==0){
		As[tx]=As[tx+8]+As[tx+16]+As[tx+24]+As[tx];
	}
	Ad[bx]=As[0];
}

__device__ __global__ void Std2(float *Ad,float *Bd){
	__device__ __shared__ float As[8],Bs[8];
	int bx=blockIdx.x;
	As[0]=Ad[bx];
	Bs[0]=Bd[bx];
	As[0]=(sqrt(As[0]/Total);
	Ad[bx]=As[0];
}

//	Here, we have a block having 512 threads.
//	Each Grid has 32 Blocks. And, we have only 1 Grid. You know.!!
//	We use shared memory the total 16KB (16384B).
//	We divide it into 32 parts. As we have 32 Blocks.
//	Each Block now has 512B of Memory (128 of floats) (32*4*128)
