#include "hip/hip_runtime.h"
/*------------------------------------------
 ---- Created By: Aditya Avinash Atluri ----
 ------- you are free to use any code ------
 ------- Submit any issues or errors -------
 -------------------------------------------*/

#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include"hip/hip_runtime.h"

__device__ __global__ void Mean1d(float *,float *);
__device__ __global__ void Meanfinald(float *);
__device__ __global__ void Std(float *,float *);
__device__ __global__ void Std1(float *,float *);

#define Block 512  	//Size of Array in Shared Memory and Number of threads in a block
#define Grid 8			//Size of Number of Blocks
#define Total 1048576	//Total number of Samples to be processed
#define Width 4096		//Total number of Samples per Grid
#define Length 16384	//Total Size of Shared Memory

int main(void){
	float B[Total/Block];
	float *A;
	A= (float *)malloc(sizeof(float)*Total);
	for(int i=0;i<Total;i++){
		A[i]=i+1;
	}
	for(int i=0;i<(Total/Block);i++){
		B[i]=0;
	}
	float *Ad,*Bd,K=1,*Kd;
	int size=Total*sizeof(float);
	int sizeb=(Total/Block)*sizeof(float);
	int sizek=sizeof(float);
	int loop=(Total/Width);
	hipMalloc((void**)&Ad,size);
	hipMalloc((void**)&Bd,sizeb);
	hipMemcpy(Ad,A,size,hipMemcpyHostToDevice);
	hipMemcpy(Bd,B,sizeb,hipMemcpyHostToDevice);
	dim3 dimBlock(Block,1);
	dim3 dimGrid(Grid,1);
	dim3 dimBlock1(Block,1);
	dim3 dimGrid1(Grid/2,1);
	for(int i=0;i<loop;i++){
		Mean1d<<<dimGrid,dimBlock>>>(Ad+(i*Width),Bd+(i*Grid));
	}
	Meanfinald<<<dimGrid1,dimBlock1>>>(Bd);
	for(int j=0;j<loop;j++){
		Std<<<dimGrid,dimBlock>>>(Ad+(j*Width));
	}
	Std1<<<dimGrid1,dimBlock1>>>(Ad,Bd);
	hipMemcpy(A,Ad,sizek,hipMemcpyDeviceToHost);
	printf("%f		%f\n",A[0],1.0);
	int quit;
	scanf("%d",&quit);
	return 0;
}


__device__ __global__ void Mean1d(float *Ad,float *Bd){
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	__device__ __shared__ float As[Block];
		As[tx]=Ad[tx+(bx*Block)];
		if(tx%8==0){
			As[tx]=(As[tx]+As[tx+1]+As[tx+2]+As[tx+3]+As[tx+4]+As[tx+5]+As[tx+6]+As[tx+7])/8;
		}
		if(tx%64==0){
			As[tx]=(As[tx]+As[tx+8]+As[tx+16]+As[tx+24]+As[tx+32]+As[tx+40]+As[tx+48]+As[tx+56])/8;
		}
		if(tx==0){
			As[tx]=(As[tx]+As[tx+64]+As[tx+128]+As[tx+192]+As[tx+256]+As[tx+320]+As[tx+384]+As[tx+448])/8;
		}
		Bd[bx]=As[0];
}

__device__ __global__ void Meanfinald(float *Bd){
	__device__ __shared__ float Bs[Block];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	Bs[tx]=Bd[tx+bx*Block];
		if(tx%8==0){
			Bs[tx]=(Bs[tx]+Bs[tx+1]+Bs[tx+2]+Bs[tx+3]+Bs[tx+4]+Bs[tx+5]+Bs[tx+6]+Bs[tx+7])/8;
		}
		if(tx%64==0){
			Bs[tx]=(Bs[tx]+Bs[tx+8]+Bs[tx+16]+Bs[tx+24]+Bs[tx+32]+Bs[tx+40]+Bs[tx+48]+Bs[tx+56])/8;
		}
		if(tx==0){
			Bs[tx]=(Bs[tx]+Bs[tx+64]+Bs[tx+128]+Bs[tx+192]+Bs[tx+256]+Bs[tx+320]+Bs[tx+384]+Bs[tx+448])/8;
		}
	Bd[bx]=Bs[0];
	Bd[0]=(Bd[0]+Bd[1]+Bd[2]+Bd[3])/4;
}

__device__ __global__ void Std(float *Ad,float *Bd){
	__device__ __shared__ float As[Block];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	As[tx]=Ad[tx+(bx*Block)];
	As[tx]=As[tx]-Bd[0];
	if(tx%8==0){
			As[tx]=(As[tx]*As[tx])+(As[tx+1]*As[tx+1])+(As[tx+2]*As[tx+2])+(As[tx+3]*As[tx+3])+(As[tx+4]*As[tx+4])+(As[tx+5]*As[tx+5])+(As[tx+6]*As[tx+6])+(As[tx+7]*As[tx+7]);
	}
	if(tx%64==0){
			As[tx]=(As[tx]+As[tx+8]+As[tx+16]+As[tx+24]+As[tx+32]+As[tx+40]+As[tx+48]+As[tx+56]);
	}
	if(tx==0){
		As[tx]=(As[tx]+As[tx+64]+As[tx+128]+As[tx+192]+As[tx+256]+As[tx+320]+As[tx+384]+As[tx+448]);
	}
	Ad[bx]=As[0];
}

__device__ __global__ void Std1(float *Ad,float *Bd){
	__device__ __shared__ float As[Block];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	if(tx%8==0){
		As[tx]=(As[tx]+As[tx+1]+As[tx+2]+As[tx+3]+As[tx+4]+As[tx+5]+As[tx+6]+As[tx+7]);
	}
	if(tx%64==0){
		As[tx]=(As[tx]+As[tx+8]+As[tx+16]+As[tx+24]+As[tx+32]+As[tx+40]+As[tx+48]+As[tx+56]);
	}
	if(tx==0){
		As[tx]=(As[tx]+As[tx+64]+As[tx+128]+As[tx+192]+As[tx+256]+As[tx+320]+As[tx+384]+As[tx+448]);
	}
	Ad[bx]=As[0];
	Ad[0]=Ad[0]+Ad[1]+Ad[2]+Ad[3];
	Ad[0]=sqrt(Ad[0]/Total);
}


//	Here, we have a block having 512 threads.
//	Each Grid has 32 Blocks. And, we have only 1 Grid. You know.!!
//	We use shared memory the total 16KB (16384B).
//	We divide it into 32 parts. As we have 32 Blocks.
//	Each Block now has 512B of Memory (128 of floats) (32*4*128)
