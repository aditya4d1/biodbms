#include "hip/hip_runtime.h"
/*------------------------------------------
 ---- Created By: Aditya Avinash Atluri ----
 ------- you are free to use any code ------
 ------- Submit any issues or errors -------
 -------------------------------------------*/

#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include"hip/hip_runtime.h"

__device__ __global__ void Mean3d(float *,float *);
__device__ __global__ void Meanfinald(float *);
__device__ __global__ void Meanfinal1d(float *);
__device__ __global__ void Std(float *,float *);
__device__ __global__ void Std1(float *);
__device__ __global__ void Std4(float *);
__device__ __global__ void Std2(float *,float *);
__device__ __global__ void Std3(float *);

#define Block 512  	//Size of Array in Shared Memory and Number of threads in a block
#define Grid 8			//Size of Number of Blocks
#define Total 1048576	//Total number of Samples per Cluster
#define Dimen 128		//Number of Clusters
#define Max 134217728	//Total number of samples
#define Width 4096		//Total number of Samples per Grid
#define Length 16384	//Total Size of Shared Memory

int main(void){
	float *A,*B;
	B=(float *)malloc(sizeof(float)*Max/Block);
	A=(float *)malloc(sizeof(float)*Max);
	for(int i=0;i<Max;i++){
		A[i]=(i+1)*0.0001;
	}
	for(int i=0;i<(Max/Block);i++){
		B[i]=0;
	}
	float *Ad,*Bd;
	int size=Max*sizeof(float);
	int sizeb=(Max/Block)*sizeof(float);
	int sizek=sizeof(float);
	int loop=(Max/Width);
	hipMalloc((void**)&Ad,size);
	hipMalloc((void**)&Bd,sizeb);
	hipMemcpy(Ad,A,size,hipMemcpyHostToDevice);
	hipMemcpy(Bd,B,sizeb,hipMemcpyHostToDevice);
	dim3 dimBlock(Block,1);
	dim3 dimGrid(Grid,1);
	dim3 dimBlock1(Block,8);
	dim3 dimGrid1(Grid,2);
	dim3 dimBlock2(2,64);
	dim3 dimGrid2(1,1);
	dim3 dimBlock3(256,1);
	dim3 dimGrid3(128,1);
	dim3 dimBlock4(256,1);
	dim3 dimGrid4(16,1);
	dim3 dimBlock5(1,1);
	dim3 dimGrid5(128,1);
	dim3 dimBlock6(16,1);
	dim3 dimGrid6(8,1);
	for(int i=0;i<loop;i++){
		Mean334d<<<dimGrid,dimBlock>>>(Ad+(i*Width),Bd+(i*Grid));
	}
	for(int i=0;i<64;i++){
		Meanfinald<<<dimGrid1,dimBlock1>>>(Bd+(i*4096));
	}
	Meanfinal1d<<<dimGrid2,dimBlock2>>>(Bd);
	for(int j=0;j<loop;j++){
		Std<<<dimGrid,dimBlock>>>(Ad+(j*Width));
	}
	Std3<<<dimGrid3,dimBlock3>>>(Ad);
	for(int i=0;i<8;i++){
		Std1<<<dimGrid4,dimBlock4>>>(Ad+i*4096);
	}
	Std4<<<dimGrid6,dimBlock6>>>(Ad);
	Std2<<<dimGrid5,dimBlock5>>>(Ad,Bd);
	hipMemcpy(A,Ad,128*sizek,hipMemcpyDeviceToHost);
	for(int i=0;i<128;i++){
		printf("%f	--	%d\n",A[i],i);
	}
	int quit;
	scanf("%d",&quit);
	return 0;
}


__device__ __global__ void Mean3d(float *Ad,float *Bd){
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	__device__ __shared__ float As[Block];
		As[tx]=Ad[tx+(bx*Block)];
		if(tx%8==0){
			As[tx]=(As[tx]+As[tx+1]+As[tx+2]+As[tx+3]+As[tx+4]+As[tx+5]+As[tx+6]+As[tx+7])/8;
		}
		if(tx%64==0){
			As[tx]=(As[tx]+As[tx+8]+As[tx+16]+As[tx+24]+As[tx+32]+As[tx+40]+As[tx+48]+As[tx+56])/8;
		}
		if(tx==0){
			As[tx]=(As[tx]+As[tx+64]+As[tx+128]+As[tx+192]+As[tx+256]+As[tx+320]+As[tx+384]+As[tx+448])/8;
		}
		Bd[bx]=As[0];
}


__device__ __global__ void Meanfinald(float *Bd){
	__device__ __shared__ float Bs[Block];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	Bs[tx]=Bd[tx+bx*Block];
		if(tx%8==0){
			Bs[tx]=(Bs[tx]+Bs[tx+1]+Bs[tx+2]+Bs[tx+3]+Bs[tx+4]+Bs[tx+5]+Bs[tx+6]+Bs[tx+7])/8;
		}
		if(tx%64==0){
			Bs[tx]=(Bs[tx]+Bs[tx+8]+Bs[tx+16]+Bs[tx+24]+Bs[tx+32]+Bs[tx+40]+Bs[tx+48]+Bs[tx+56])/8;
		}
		if(tx==0){
			Bs[tx]=(Bs[tx]+Bs[tx+64]+Bs[tx+128]+Bs[tx+192]+Bs[tx+256]+Bs[tx+320]+Bs[tx+384]+Bs[tx+448])/8;
		}
	Bd[bx]=Bs[0];
	int ty=threadIdx.y;
	int by=blockIdx.y;
	__device__ __shared__ float Cs[8];
	Cs[ty]=Bd[ty+by*8];
	if(ty==0){
		Cs[ty]=(Cs[ty]+Cs[ty+1]+Cs[ty+2]+Cs[ty+3]+Cs[ty+4]+Cs[ty+5]+Cs[ty+6]+Cs[ty+7])/8;
	}
	Bd[by]=Cs[0];
}


__device__ __global__ void Meanfinal1d(float *Bd){
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	Bd[tx+ty*2]=Bd[tx+ty*4096];
}


__device__ __global__ void Std(float *Ad,float *Bd){
	__device__ __shared__ float As[Block];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	As[tx]=Ad[tx+(bx*Block)];
	As[tx]=As[tx]-Bd[0];
	if(tx%8==0){
			As[tx]=(As[tx]*As[tx])+(As[tx+1]*As[tx+1])+(As[tx+2]*As[tx+2])+(As[tx+3]*As[tx+3])+(As[tx+4]*As[tx+4])+(As[tx+5]*As[tx+5])+(As[tx+6]*As[tx+6])+(As[tx+7]*As[tx+7]);
	}
	if(tx%64==0){
			As[tx]=(As[tx]+As[tx+8]+As[tx+16]+As[tx+24]+As[tx+32]+As[tx+40]+As[tx+48]+As[tx+56]);
	}
	if(tx==0){
		As[tx]=(As[tx]+As[tx+64]+As[tx+128]+As[tx+192]+As[tx+256]+As[tx+320]+As[tx+384]+As[tx+448]);
	}
	Ad[bx]=As[0];
	Ad[0]=(Ad[0]+Ad[1]+Ad[2]+Ad[3]+Ad[4]+Ad[5]+Ad[6]+Ad[7])/8;
}


__device__ __global__ void Std1(float *Ad){
	__device__ __shared__ float As[256];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	As[tx]=Ad[tx+bx*256];
	if(tx%8==0){
		As[tx]=(As[tx]+As[tx+1]+As[tx+2]+As[tx+3]+As[tx+4]+As[tx+5]+As[tx+6]+As[tx+7]);
	}
	if(tx%64==0){
		As[tx]=(As[tx]+As[tx+8]+As[tx+16]+As[tx+24]+As[tx+32]+As[tx+40]+As[tx+48]+As[tx+56]);
	}
	if(tx%256==0){
		As[tx]=(As[tx]+As[tx+64]+As[tx+128]+As[tx+192]);
	}
	Ad[bx]=As[0];
}


__device__ __global__ void Std3(float *Ad){
	int tx=threadIdx.x;
	int by=blockIdx.y;
	int bx=blockIdx.x;
	Ad[tx+bx*256]=Ad[tx*4096+bx*1048576];
}


__device__ __global__ void Std4(float *Ad){
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	Ad[tx+bx*16]=Ad[tx+bx*4096];
}


__device__ __global__ void Std2(float *Ad,float *Bd){
	__device__ __shared__ float As[1],Bs[1];
	int bx=blockIdx.x;
	As[0]=Ad[bx];
	Bs[0]=Bd[bx];
	As[0]=sqrt(As[0]/Total);
	Ad[bx]=As[0];
}


//	Here, we have a block having 512 threads.
//	Each Grid has 32 Blocks. And, we have only 1 Grid. You know.!!
//	We use shared memory the total 16KB (16384B).
//	We divide it into 32 parts. As we have 32 Blocks.
//	Each Block now has 512B of Memory (128 of floats) (32*4*128)
