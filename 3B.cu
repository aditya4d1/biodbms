#include "hip/hip_runtime.h"
/*------------------------------------------
 ---- Created By: Aditya Avinash Atluri ----
 ------- you are free to use any code ------
 ------- Submit any issues or errors -------
 -------------------------------------------*/

#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include"hip/hip_runtime.h"

__device__ __global__ void Mean3b(float *,float *);
__device__ __global__ void Meanfinald(float *);
__device__ __global__ void Meanfinal1d(float *);
__device__ __global__ void Std(float *,float *);
__device__ __global__ void Std1(float *);
__device__ __global__ void Std2(float *,float *);

#define Block 512  	//Size of Array in Shared Memory and Number of threads in a block
#define Grid 8			//Size of Number of Blocks
#define Dimen 128		//Number of Clusters
#define Total 16384		//Total number of Samples per Cluster
#define Max 2097152		//Total number of Samples
#define Width 4096		//Total number of Samples per Grid
#define Length 16384	//Total Size of Shared Memory

int main(void){
	float *A,B[Max/Block];
	A=(float *)malloc(Max*sizeof(float));
	for(int i=0;i<Max;i++){
		A[i]=(0.001)*i;
	}
	for(int i=0;i<(Max/Block);i++){
		B[i]=0;
	}
	float *Ad,*Bd;
	int size=Max*sizeof(float);
	int sizeb=(Max/Block)*sizeof(float);
	int sizek=sizeof(float);
	int loop=(Max/Width);
	hipMalloc((void**)&Ad,size);
	hipMalloc((void**)&Bd,sizeb);
	hipMemcpy(Ad,A,size,hipMemcpyHostToDevice);
	hipMemcpy(Bd,B,sizeb,hipMemcpyHostToDevice);
	dim3 dimBlock(Block,1);
	dim3 dimGrid(Grid,1);
	dim3 dimBlock1(32,1);
	dim3 dimGrid1(32,1);
	dim3 dimBlock2(8,1);
	dim3 dimGrid2(32,1);
	dim3 dimBlock3(32,1);
	dim3 dimGrid3(4,1);
	for(int i=0;i<loop;i++){
		Mean3b<<<dimGrid,dimBlock>>>(Ad+(i*Width),Bd+(i*Grid));
	}
	for(int i=0;i<4;i++){
		Meanfinald<<<dimGrid1,dimBlock1>>>(Bd+(i*1024));
	}
	Meanfinal1d<<<dimGrid2,dimBlock2>>>(Bd);
	for(int j=0;j<loop;j++){
		Std<<<dimGrid,dimBlock>>>(Ad+(j*Width),Bd);
	}
	for(int i=0;i<4;i++){
		Std1<<<dimGrid2,dimBlock2>>>(Ad+(i*524288));
	}
	Std2<<<dimGrid3,dimBlock3>>>(Ad,Bd);
	hipMemcpy(A,Ad,128*sizek,hipMemcpyDeviceToHost);
	for(int i=0;i<128;i++){
		printf("%f	-- SD	%d\n",A[i],i);
	}
	int quit;
	scanf("%d",&quit);
	return 0;
}


__device__ __global__ void Mean3b(float *Ad,float *Bd){
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	__device__ __shared__ float As[Block];
		As[tx]=Ad[tx+(bx*Block)];
		if(tx%8==0){
			As[tx]=(As[tx]+As[tx+1]+As[tx+2]+As[tx+3]+As[tx+4]+As[tx+5]+As[tx+6]+As[tx+7])/8;
		}
		if(tx%64==0){
			As[tx]=(As[tx]+As[tx+8]+As[tx+16]+As[tx+24]+As[tx+32]+As[tx+40]+As[tx+48]+As[tx+56])/8;
		}
		if(tx==0){
			As[tx]=(As[tx]+As[tx+64]+As[tx+128]+As[tx+192]+As[tx+256]+As[tx+320]+As[tx+384]+As[tx+448])/8;
		}
		Bd[bx]=As[0];
}

__device__ __global__ void Meanfinald(float *Bd){
	__device__ __shared__ float Bs[32];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	Bs[tx]=Bd[tx+bx*32];
	if(tx%8==0){
		Bs[tx]=(Bs[tx]+Bs[tx+1]+Bs[tx+2]+Bs[tx+3]+Bs[tx+4]+Bs[tx+5]+Bs[tx+6]+Bs[tx+7])/8;
	}
	if(tx%32==0){
		Bs[tx]=(Bs[tx]+Bs[tx+8]+Bs[tx+16]+Bs[tx+24])/4;
	}
	Bd[bx]=Bs[0];
}

__device__ __global__ void Meanfinal1d(float *Bd){
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	Bd[tx+bx*32]=Bd[tx+bx*1024];
}

__device__ __global__ void Std(float *Ad,float *Bd){
	__device__ __shared__ float As[Block];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	As[tx]=Ad[tx+(bx*Block)];
	As[tx]=As[tx]-Bd[0];
	if(tx%8==0){
			As[tx]=(As[tx]*As[tx])+(As[tx+1]*As[tx+1])+(As[tx+2]*As[tx+2])+(As[tx+3]*As[tx+3])+(As[tx+4]*As[tx+4])+(As[tx+5]*As[tx+5])+(As[tx+6]*As[tx+6])+(As[tx+7]*As[tx+7]);
	}
	if(tx%64==0){
			As[tx]=(As[tx]+As[tx+8]+As[tx+16]+As[tx+24]+As[tx+32]+As[tx+40]+As[tx+48]+As[tx+56]);
	}
	if(tx==0){
		As[tx]=(As[tx]+As[tx+64]+As[tx+128]+As[tx+192]+As[tx+256]+As[tx+320]+As[tx+384]+As[tx+448]);
	}
	Ad[bx]=As[0];
}

__device__ __global__ void Std1(float *Ad){
	__device__ __shared__ float As[32];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	for(int i=0;i<4;i++){
		As[tx+i*8]=Ad[tx+i*4096+bx*16384];
	}
	if(tx%8==0){
		As[tx]=(As[tx]+As[tx+1]+As[tx+2]+As[tx+3]+As[tx+4]+As[tx+5]+As[tx+6]+As[tx+7]);
	}
	if(tx%32==0){
		As[tx]=(As[tx]+As[tx+8]+As[tx+16]+As[tx+24]);
	}
	Ad[bx]=As[0];
}


__device__ __global__ void Std2(float *Ad,float*Bd){
	__device__ __shared__ float As[128],Bs[128];
	int tx=threadIdx.x;
	int bx=blockIdx.x;
	As[tx]=Ad[tx+bx*524288];
	Bs[tx]=Bd[tx+bx*32];
	{
		As[tx]=sqrt(As[tx]/Total);
	}
	Ad[tx+bx*32]=As[tx];
}

//	Here, we have a block having 512 threads.
//	Each Grid has 32 Blocks. And, we have only 1 Grid. You know.!!
//	We use shared memory the total 16KB (16384B).
//	We divide it into 32 parts. As we have 32 Blocks.
//	Each Block now has 512B of Memory (128 of floats) (32*4*128)
